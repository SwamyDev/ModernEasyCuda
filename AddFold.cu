#include "hip/hip_runtime.h"
#include "AddFold.hpp"

__global__ void add(std::size_t  n, const float *x, float *y) {
  std::size_t  index = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t  stride = blockDim.x * gridDim.x;
  for (auto i = index; i < n; i += stride) y[i] = x[i] + y[i];
}

void add_on_gpu(std::size_t n, const float *src, float *dst) {
  std::size_t blockSize = 256;
  std::size_t numBlocks = (n + blockSize - 1)/blockSize;
  add<<<numBlocks, blockSize>>> (n, src, dst);

  hipDeviceSynchronize();
}